
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_fp16.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

typedef half2 ftype;

#define BENCH_TIMES 1
#define WARMUP for (int warm_up = 0; warm_up < 0; warm_up++)
#define BENCH_TEST for (int test_round = 0; test_round < BENCH_TIMES; test_round++)

int main(int argc, char** argv) {
    long long n = atoi(argv[1]);
    long long sig_size = 1 << n;
    ftype *h_idata = (ftype *)malloc(sig_size * sizeof(ftype));
    ftype *d_idata;
    ftype *d_odata;
    hipMalloc(&d_idata, sizeof(ftype) * sig_size);
    hipMalloc(&d_odata, sizeof(ftype) * sig_size);
    hipfftHandle plan;
    hipfftResult r;
    r = hipfftCreate(&plan);
    assert(r == HIPFFT_SUCCESS);
    size_t ws = 0;
    r = hipfftXtMakePlanMany(plan, 1, &sig_size, NULL, 1, 1, HIP_C_16F, NULL, 1,
                            1, HIP_C_16F, 1, &ws, HIP_C_16F);
    assert(r == HIPFFT_SUCCESS);
    WARMUP
    {r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD);}  // warm-up
    assert(r == HIPFFT_SUCCESS);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    BENCH_TEST
    {r = hipfftXtExec(plan, d_idata, d_odata, HIPFFT_FORWARD);}
    assert(r == HIPFFT_SUCCESS);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    et/=BENCH_TIMES;
    // printf("forward FFT time for %lld samples: %fms\n", sig_size, et);
    double gflops = (5.0 * sig_size * 1e-6 * log2(sig_size) / et);
    printf("1d,16,%lld,%f,%lf,%lld\n", sig_size, et, gflops, sig_size);

    return 0;
}