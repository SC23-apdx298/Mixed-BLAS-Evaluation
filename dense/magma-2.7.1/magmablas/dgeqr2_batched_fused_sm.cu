#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date February 2023

       @author Ahmad Abdelfattah
       @author Azzam Haidar

       @generated from magmablas/zgeqr2_batched_fused_sm.cu, normal z -> d, Wed Feb 22 15:21:26 2023
*/

#include <hip/hip_runtime.h>    // for CUDA_VERSION
#include "magma_internal.h"
#include "magma_templates.h"
#include "dgeqr2_batched_fused.cuh"
#include "batched_kernel_param.h"

#define PRECISION_d

////////////////////////////////////////////////////////////////////////////////
__global__
void
dgeqr2_fused_sm_kernel_batched(
    int M, int N,
    double **dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t ldda,
    double **dtau_array, magma_int_t taui,
    magma_int_t *info_array, magma_int_t batchCount)
{
    extern __shared__ double zdata[];
    const int tx  = threadIdx.x;
    const int ty  = threadIdx.y;
    const int ntx = blockDim.x;
    const int nty = blockDim.y;
    const int batchid = blockIdx.x * nty + ty;
    if(batchid >= batchCount) return;

    const int slda  = SLDA(M);
    double* dA   = dA_array[batchid] + Aj * ldda + Ai;
    double* dtau = dtau_array[batchid] + taui;
    magma_int_t* info = &info_array[batchid];

    // shared memory pointers
    double* sA    = (double*)(zdata);
    double* sY    = sA   + (nty * slda * N);
    double* stau  = sY   + (nty * N);
    double* sTmp  = stau + nty * N;
    sA    += ty * slda * N;
    sY    += ty * N;
    stau  += ty * N;
    sTmp  += ty * ntx;
    double* snorm = (double*) (sTmp); // must be set after offsetting w.r.t. ty

    double alpha, tau, tmp, scale = MAGMA_D_ZERO;
    double norm = MAGMA_D_ZERO, beta;

    if( tx == 0 ){
        (*info) = 0;
    }

    // init tau
    if(tx < N) {
        stau[tx] = MAGMA_D_ZERO;
    }

    // read
    for(int j = 0; j < N; j++){
        for(int i = tx; i < M; i+=ntx) {
            sA(i,j) = dA[ j * ldda + i ];
        }
    }
    __syncthreads();

    for(int j = 0; j < N; j++){
        alpha = sA(j,j);

        dgeqr2_compute_norm(M-j, &sA(j,j), snorm, tx, ntx);
        // there is a sync at the end of dgeqr2_compute_norm

        norm = sqrt(snorm[0]);
        beta = -copysign(norm, real(alpha));
        scale = MAGMA_D_DIV( MAGMA_D_ONE,  alpha - MAGMA_D_MAKE(beta, 0));
        tau = MAGMA_D_MAKE( (beta - real(alpha)) / beta, -imag(alpha) / beta );

        if(tx == 0) {
            stau[j] = tau;
            sA(j,j) = MAGMA_D_ONE;
        }

        // scale the current column below the diagonal
        for(int i = (tx+j+1); i < M; i+=ntx) {
            sA(i,j) *= scale;
        }
        __syncthreads();

        // copy the first portion of the column into tmp
        // since M > N and ntx >= N, this portion must
        // have the diagonal
        tmp = (tx == j) ? MAGMA_D_MAKE(beta, MAGMA_D_ZERO) : sA(tx, j);

        // write the column into global memory
        dA[j * ldda + tx] = tmp;
        for(int i = tx+ntx; i < M; i+=ntx) {
            dA[ j * ldda + i ] = sA(i, j);
        }

        // now compute (I - tau * v * v') A
        // first: y = tau * v' * A (row vector)
        dgeqr2_compute_vtA_device(M, N, j, sA, slda, sY, tau, sTmp, tx, ntx);
        __syncthreads();

        // now compute: A = A - v * y
        for(int jj = j+1; jj < N; jj++){
            for(int i = tx+j; i < M; i+=ntx) {
                sA(i,jj) -= sA(i,j) * sY[jj];
            }
        }
        __syncthreads();
    }

    // write tau and the last column
    if(tx < N) {
        dtau[tx] = stau[tx];
    }
}

////////////////////////////////////////////////////////////////////////////////
extern "C" magma_int_t
magma_dgeqr2_fused_sm_batched(
    magma_int_t m, magma_int_t n,
    double** dA_array, magma_int_t Ai, magma_int_t Aj, magma_int_t ldda,
    double **dtau_array, magma_int_t taui,
    magma_int_t* info_array, magma_int_t nthreads, magma_int_t check_launch_only,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t arginfo = 0;
    magma_device_t device;
    magma_getdevice( &device );

    if (m < 0)
        arginfo = -1;
    else if (n < 0)
        arginfo = -2;
    else if (ldda < max(1,m))
        arginfo = -4;

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }

    /* Quick return if possible */
    if (m == 0 || n == 0)
        return arginfo;

    // disable this kernel for n > 8
    if( m < n || n > 8) return -100;

    nthreads = min(nthreads, m);

    const magma_int_t ntcol = 1;
    magma_int_t shmem = ( SLDA(m) * n * sizeof(double) );
    shmem            += ( n        * sizeof(double) );  // sY
    shmem            += ( n        * sizeof(double) );  // stau
    shmem            += ( nthreads * sizeof(double) );  // used for snorm and for computing v' * A
    shmem            *= ntcol;
    magma_int_t gridx = magma_ceildiv(batchCount, ntcol);
    dim3 grid(gridx, 1, 1);
    dim3 threads( nthreads, ntcol, 1);

    // get max. dynamic shared memory on the GPU
    int nthreads_max, shmem_max = 0;
    hipDeviceGetAttribute (&nthreads_max, hipDeviceAttributeMaxThreadsPerBlock, device);
    #if CUDA_VERSION >= 9000
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeSharedMemPerBlockOptin, device);
    if (shmem <= shmem_max) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(dgeqr2_fused_sm_kernel_batched), hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
    }
    #else
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    #endif    // CUDA_VERSION >= 9000

    magma_int_t total_threads = nthreads * ntcol;
    if ( total_threads > nthreads_max || shmem > shmem_max ) {
        // printf("error: kernel %s requires too many threads or too much shared memory\n", __func__);
        arginfo = -100;
        return arginfo;
    }

    if( check_launch_only == 1 ) return arginfo;

    void *kernel_args[] = {&m, &n, &dA_array, &Ai, &Aj, &ldda, &dtau_array, &taui, &info_array, &batchCount};
    hipError_t e = hipLaunchKernel((void*)dgeqr2_fused_sm_kernel_batched, grid, threads, kernel_args, shmem, queue->hip_stream());
    if( e != hipSuccess ) {
        // printf("error in %s : failed to launch kernel %s\n", __func__, hipGetErrorString(e));
        arginfo = -100;
    }

    return arginfo;
}
