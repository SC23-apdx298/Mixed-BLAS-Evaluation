#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date February 2023

       @generated from magmablas/zhetrs.cu, normal z -> s, Wed Feb 22 15:21:24 2023

       @author Stan Tomov
*/

#include "magma_internal.h"
#include "magma_templates.h"

#define REAL

#define dA(i_, j_) (dA + (i_) + (j_)*ldda)
#define dB(i_, j_) (dB + (i_) + (j_)*lddb)

#ifdef COMPLEX
void magmablas_slacgv( magma_int_t n, float *x, magma_int_t incx, magma_queue_t queue );
#endif

__global__ void
magma_sger_1(
    int n, int k, int kp, int nrhs, 
    float *A, float *B, int lddb)
{
    int tx  = threadIdx.x + 64 * blockIdx.x;

    if (k+1+tx < n)
        if (k!=kp && k+1+tx == kp)
            // if k <-> kp pivoting, B[k] holds the result for B[kp]
            B[k]  -= A[kp]*B[kp];
        else
            B[k+1+tx] -= A[k+1+tx]*B[kp];
}

__global__ void
magma_sswap_scal(
    int k, int kp, int nrhs, 
    float *A, float *B, int lddb)
{
    float tmp;
    if (k != kp){
        tmp   = B[k];
        B[k]  = B[kp];
        B[kp] = tmp;
    }
    B[k] *= MAGMA_S_DIV(MAGMA_S_ONE, A[k]);
}

__global__ void
magma_sger_2(
    int n, int k, int kp, int nrhs,
    float *A, int ldda, float *B, int lddb)
{
    int tx  = threadIdx.x + 64 * blockIdx.x;

    if (k+2+tx < n)
        if (k+1!=kp && k+2+tx == kp)
            // if k+1 <-> kp pivoting, B[k+1] holds the result for B[kp]
            B[k+1]  -= A[kp]*B[k] + A[kp+ldda]*B[kp];
        else
            B[k+2+tx] -= A[k+2+tx]*B[k] + A[k+2+tx+ldda]*B[kp];
}

__global__ void
magma_sswap_scal_inverseblock_lower(
    int k, int kp, int nrhs, 
    float *dA, int ldda, float *dB, int lddb)
{
    int tx  = threadIdx.x;

    float tmp;
    if (k+1 != kp){
        tmp         = *dB(k+1,tx);
        *dB(k+1,tx) = *dB(kp ,tx);
        *dB( kp,tx) = tmp;
    }

    float AKM1K = *dA(1,0);
    float AKM1  = MAGMA_S_DIV(*dA(0,0), MAGMA_S_CONJ( AKM1K ) );
    float AK    = MAGMA_S_DIV(*dA(1,1), AKM1K );
    float DENOM = AKM1*AK - MAGMA_S_ONE;

    float  BKM1 = MAGMA_S_DIV( *dB(k,tx),  MAGMA_S_CONJ(AKM1K));
    float  BK   = MAGMA_S_DIV( *dB(k+1,tx), AKM1K );

    *dB(k,tx) = MAGMA_S_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(k+1,tx) = MAGMA_S_DIV( AKM1*BK-BKM1,  DENOM );
}

// This kernel scales the array B by 1/alpha.
// The kernel is called on one thread block with thread equal the 
// length of B, so that each thread scales just one element of B.
__global__ void
magmablas_sscal_inverse(
    float *alpha, 
    float *B, int ldb)
{
    int tx  = threadIdx.x;

    float scale = MAGMA_S_DIV(MAGMA_S_ONE, *alpha);
    B[tx*ldb] *= scale;
}

// Multiply array dB of size 2 by the inverse of the 2x2 diagonal block at dA.
// This is a batch operation where each thread is doing one multiplication.
__global__ void
magmablas_sscal_inverseblock_upper(
    const float *dA, int ldda, 
    float *dB, int lddb)
{
    int tx  = threadIdx.x;
    
    float AKM1K = *dA(0,1);
    float AKM1  = MAGMA_S_DIV(*dA(0,0), AKM1K);
    float AK    = MAGMA_S_DIV(*dA(1,1), MAGMA_S_CONJ( AKM1K ));
    float DENOM = AKM1*AK - MAGMA_S_ONE;

    float  BKM1 = MAGMA_S_DIV( *dB(0,tx), AKM1K);
    float  BK   = MAGMA_S_DIV( *dB(1,tx), MAGMA_S_CONJ(AKM1K) );

    *dB(0,tx) = MAGMA_S_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(1,tx) = MAGMA_S_DIV( AKM1*BK-BKM1,  DENOM );
}

__global__ void
magmablas_sscal_inverseblock_lower(
    const float *dA, int ldda,
    float *dB, int lddb)
{
    int tx  = threadIdx.x;

    float AKM1K = *dA(1,0);
    float AKM1  = MAGMA_S_DIV(*dA(0,0), MAGMA_S_CONJ( AKM1K ) );
    float AK    = MAGMA_S_DIV(*dA(1,1), AKM1K );
    float DENOM = AKM1*AK - MAGMA_S_ONE;

    float  BKM1 = MAGMA_S_DIV( *dB(0,tx),  MAGMA_S_CONJ(AKM1K));
    float  BK   = MAGMA_S_DIV( *dB(1,tx), AKM1K );

    *dB(0,tx) = MAGMA_S_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(1,tx) = MAGMA_S_DIV( AKM1*BK-BKM1,  DENOM );
}


/***************************************************************************//**
    Purpose
    -------
    SSYTRS solves a system of linear equations dA*dX = dB with a real
    symmetric matrix dA using the factorization dA = dU*dD*dU**H or
    dA = dL*dD*dL**H computed by SSYTRF_GPU.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t 
            Specifies whether the details of the factorization are stored
            as an upper or lower triangular matrix.
            = MagmaUpper:  Upper triangular, form is A = U*D*U**H;
            = MagmaLower:  Lower triangular, form is A = L*D*L**H.
    
    @param[in]
    n       INTEGER
            The order of the matrix dA.  N >= 0.
    
    @param[in]
    nrhs    INTEGER
            The number of right hand sides, i.e., the number of columns
            of the matrix dB.  NRHS >= 0.
    
    @param[in]
    dA      REAL array, dimension (LDA,N)
            The block diagonal matrix D and the multipliers used to
            obtain the factor U or L as computed by SSYTRF_GPU.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,N).
    
    @param[in]
    ipiv    INTEGER array, dimension (N)
            Details of the interchanges and the block structure of D
            as determined by SSYTRF.
    
    @param[in,out]
    dB      REAL array, dimension (LDDB, NRHS)
            On entry, the right hand side matrix dB.
            On exit, the solution matrix dX.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,N).
    
    @param[out]
    info    INTEGER
            = 0:  successful exit
            < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_hetrs
*******************************************************************************/
extern "C" magma_int_t
magma_ssytrs_gpu(
    magma_uplo_t uplo, magma_int_t n, magma_int_t nrhs,
    float *dA, magma_int_t ldda,
    magma_int_t *ipiv,
    float *dB, magma_int_t lddb,
    magma_int_t *info, 
    magma_queue_t queue )
{
    /* Constants */
    const float c_one     = MAGMA_S_ONE;
    const float c_neg_one = MAGMA_S_NEG_ONE;

    /* Local variables */
    int k, kp;
    bool upper = (uplo == MagmaUpper);

    /* Test the input parameters. */
    *info = 0;
    if ( ! upper && uplo != MagmaLower ) {
        *info = -1;
    } else if ( n < 0 ) {
        *info = -2;
    } else if ( nrhs < 0 ) {
        *info = -3;
    } else if ( ldda < max( 1, n ) ) {
        *info = -5;
    } else if ( lddb < max( 1, n ) ) {
        *info = -8;
    }
    
    if ( *info != 0 ) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Quick return if possible */
    if (n == 0 || nrhs == 0) {
        return *info;
    }
    
    if (upper) {
        /* Solve A*X = B, where A = U*D*U'.
           First solve U*D*X = B, overwriting B with X.
           K is the main loop index, decreasing from N to 1 in steps of
           1 or 2, depending on the size of the diagonal blocks.      */
        for(k = n-1; k > -1; k--) {
            if ( ipiv[k] > 0 ) {
                /* 1 x 1 diagonal block.
                   Interchange rows k and ipiv(k). */
                kp = ipiv[k]-1;
                if ( kp != k )
                    magma_sswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);

                /* Multiply by inv(U(K)), where U(K) is the transformation
                   stored in column K of A. */
                magma_sger(k, nrhs, c_neg_one, dA(0,k), 1, dB(k,0), lddb, dB(0,0), lddb, queue);

                /* Multiply by the inverse of the diagonal block. */
                magmablas_sscal_inverse<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), dB(k,0), lddb);
            } 
            else {
                /* 2 x 2 diagonal block
                   Interchange rows K-1 and -IPIV(K). */
                kp = -ipiv[k]-1;
                if ( kp != k-1 )
                    magma_sswap(nrhs, dB(k-1,0), lddb, dB(kp,0), lddb, queue);

                /* Multiply by inv(U(K)), where U(K) is the transformation
                   stored in columns K-1 and K of A. */
                magma_sger(k-1, nrhs, c_neg_one, dA(0,k  ), 1, dB(k  ,0), lddb, dB, lddb, queue);
                magma_sger(k-1, nrhs, c_neg_one, dA(0,k-1), 1, dB(k-1,0), lddb, dB, lddb, queue);

                /* Multiply by the inverse of the diagonal block. */
                magmablas_sscal_inverseblock_upper<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k-1,k-1), ldda, dB(k-1,0), lddb);

                /* reduce k once more for the 2 x 2 block */
                k--;
            }
        }

        /* Next solve U'*X = B, overwriting B with X.
           K is the main loop index, increasing from 1 to N in steps of
           1 or 2, depending on the size of the diagonal blocks.  */
        for(k=0; k<n; k++) {
            if ( ipiv[k] > 0) {
                /* 1 x 1 diagonal block
                   Multiply by inv(U'(K)), where U(K) is the transformation
                   stored in column K of A. */
                if (k > 0) {
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k), 1, c_one, dB(k, 0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv( nrhs, dB(k,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if( kp != k )
                    magma_sswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);
            }
            else {
                /* 2 x 2 diagonal block
                   Multiply by inv(U'(K+1)), where U(K+1) is the transformation
                   stored in columns K and K+1 of A. */
                if (k > 0) {
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k), 1, c_one, dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif

                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k+1,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k+1), 1, c_one, dB(k+1, 0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k+1,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and -IPIV(K). */
                kp = -ipiv[k]-1;
                if ( kp != k )
                    magma_sswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);
            
                /* increase k one more for the 2 x 2 block */
                k++;
            }
        }    
    } else {
        /*  Solve A*X = B, where A = L*D*L'.
            First solve L*D*X = B, overwriting B with X.
    
            K is the main loop index, increasing from 1 to N in steps of
            1 or 2, depending on the size of the diagonal blocks. */
        for(k=0; k<n; k++) {
            if ( ipiv[k] > 0) {
                /* 1 x 1 diagonal block 
                   Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if (0){
                if ( kp != k )
                    magma_sswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);
                
                /* Multiply by inv(L(K)), where L(K) is the transformation
                   stored in column K of A.   */
                if (k < n-1)
                    magma_sger(n-k-1, nrhs, c_neg_one, dA(k+1, k), 1, dB(k,0), lddb,
                                dB(k+1, 0), lddb, queue);
                
                /* Multiply by the inverse of the diagonal block. */
                magmablas_sscal_inverse<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), dB(k,0), lddb);
                }     
                else {
                    magma_sger_1<<<magma_ceildiv(n-k-1,64), 64, 0, queue->hip_stream()>>>
                        (n, k, kp, nrhs, dA(0,k), dB, lddb);
                    magma_sswap_scal<<<1, nrhs, 0, queue->hip_stream()>>>
                        (k, kp, nrhs, dA(0,k), dB, lddb);
                }
            }
            else {
                /*  2 x 2 diagonal block
                    Interchange rows K+1 and -IPIV(K). */
                kp = -ipiv[k]-1;
                if (0) {
                if ( kp != k+1 )
                    magma_sswap(nrhs, dB(k+1,0), lddb, dB(kp,0), lddb, queue);

                /* Multiply by inv(L(K)), where L(K) is the transformation
                   stored in columns K and K+1 of A. */
                if ( k < n-2 ) {
                    magma_sger(n-k-2, nrhs, c_neg_one, dA(k+2,k), 1, dB(k,0), lddb,
                                dB(k+2,0), lddb, queue);
                    magma_sger(n-k-2, nrhs, c_neg_one, dA(k+2,k+1), 1, dB(k+1,0), lddb, 
                                dB(k+2,0), lddb, queue);
                }
            
                /* Multiply by the inverse of the diagonal block. */
                magmablas_sscal_inverseblock_lower<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), ldda, dB(k,0), lddb);
                }
                else {
                    magma_sger_2<<<magma_ceildiv(n-k-2,64), 64, 0, queue->hip_stream()>>>
                        (n, k, kp, nrhs, dA(0,k), ldda, dB, lddb);
                    magma_sswap_scal_inverseblock_lower<<<1, nrhs, 0, queue->hip_stream()>>>
                        (k, kp, nrhs, dA(k,k), ldda, dB(0, 0), lddb);
                }

                /* increase k one more for the 2 x 2 block */
                k++;
            }
        }
        
        /* Next solve L'*X = B, overwriting B with X.
           K is the main loop index, decreasing from N to 1 in steps of
           1 or 2, depending on the size of the diagonal blocks. */
        for(k = n-1; k > -1; k--) {
            if ( ipiv[k] > 0 ) {
                /* 1 x 1 diagonal block.
                   Multiply by inv(L'(K)), where L(K) is the transformation
                   stored in column K of A. */
                if (1){
                if ( k < n-1 ) {
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k), 1, c_one,
                                dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if ( kp!=k )
                    magma_sswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);
                }
                else {
                }
            }
            else {
                /*  2 x 2 diagonal block
                    Multiply by inv(L'(K-1)), where L(K-1) is the transformation
                    stored in columns K-1 and K of A. */
                if ( k < n-1 ) {
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k), 1, c_one,
                                dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k,0), lddb, queue);
                    #endif

                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k-1,0), lddb, queue);
                    #endif
                    magma_sgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k-1), 1, c_one,
                                dB(k-1,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_slacgv(nrhs, dB(k-1,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and -IPIV(K).*/
                kp = -ipiv[k]-1;
                if ( kp != k )
                    magma_sswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);

                /* reduce k once more for the 2 x 2 block */
                k--;
            }
        }

    }
    return *info;
}
