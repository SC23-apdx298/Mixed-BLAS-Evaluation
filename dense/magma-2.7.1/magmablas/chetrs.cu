#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date February 2023

       @generated from magmablas/zhetrs.cu, normal z -> c, Wed Feb 22 15:21:24 2023

       @author Stan Tomov
*/

#include "magma_internal.h"
#include "magma_templates.h"

#define COMPLEX

#define dA(i_, j_) (dA + (i_) + (j_)*ldda)
#define dB(i_, j_) (dB + (i_) + (j_)*lddb)

#ifdef COMPLEX
void magmablas_clacgv( magma_int_t n, magmaFloatComplex *x, magma_int_t incx, magma_queue_t queue );
#endif

__global__ void
magma_cgeru_1(
    int n, int k, int kp, int nrhs, 
    magmaFloatComplex *A, magmaFloatComplex *B, int lddb)
{
    int tx  = threadIdx.x + 64 * blockIdx.x;

    if (k+1+tx < n)
        if (k!=kp && k+1+tx == kp)
            // if k <-> kp pivoting, B[k] holds the result for B[kp]
            B[k]  -= A[kp]*B[kp];
        else
            B[k+1+tx] -= A[k+1+tx]*B[kp];
}

__global__ void
magma_cswap_scal(
    int k, int kp, int nrhs, 
    magmaFloatComplex *A, magmaFloatComplex *B, int lddb)
{
    magmaFloatComplex tmp;
    if (k != kp){
        tmp   = B[k];
        B[k]  = B[kp];
        B[kp] = tmp;
    }
    B[k] *= MAGMA_C_DIV(MAGMA_C_ONE, A[k]);
}

__global__ void
magma_cgeru_2(
    int n, int k, int kp, int nrhs,
    magmaFloatComplex *A, int ldda, magmaFloatComplex *B, int lddb)
{
    int tx  = threadIdx.x + 64 * blockIdx.x;

    if (k+2+tx < n)
        if (k+1!=kp && k+2+tx == kp)
            // if k+1 <-> kp pivoting, B[k+1] holds the result for B[kp]
            B[k+1]  -= A[kp]*B[k] + A[kp+ldda]*B[kp];
        else
            B[k+2+tx] -= A[k+2+tx]*B[k] + A[k+2+tx+ldda]*B[kp];
}

__global__ void
magma_cswap_scal_inverseblock_lower(
    int k, int kp, int nrhs, 
    magmaFloatComplex *dA, int ldda, magmaFloatComplex *dB, int lddb)
{
    int tx  = threadIdx.x;

    magmaFloatComplex tmp;
    if (k+1 != kp){
        tmp         = *dB(k+1,tx);
        *dB(k+1,tx) = *dB(kp ,tx);
        *dB( kp,tx) = tmp;
    }

    magmaFloatComplex AKM1K = *dA(1,0);
    magmaFloatComplex AKM1  = MAGMA_C_DIV(*dA(0,0), MAGMA_C_CONJ( AKM1K ) );
    magmaFloatComplex AK    = MAGMA_C_DIV(*dA(1,1), AKM1K );
    magmaFloatComplex DENOM = AKM1*AK - MAGMA_C_ONE;

    magmaFloatComplex  BKM1 = MAGMA_C_DIV( *dB(k,tx),  MAGMA_C_CONJ(AKM1K));
    magmaFloatComplex  BK   = MAGMA_C_DIV( *dB(k+1,tx), AKM1K );

    *dB(k,tx) = MAGMA_C_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(k+1,tx) = MAGMA_C_DIV( AKM1*BK-BKM1,  DENOM );
}

// This kernel scales the array B by 1/alpha.
// The kernel is called on one thread block with thread equal the 
// length of B, so that each thread scales just one element of B.
__global__ void
magmablas_csscal_inverse(
    magmaFloatComplex *alpha, 
    magmaFloatComplex *B, int ldb)
{
    int tx  = threadIdx.x;

    magmaFloatComplex scale = MAGMA_C_DIV(MAGMA_C_ONE, *alpha);
    B[tx*ldb] *= scale;
}

// Multiply array dB of size 2 by the inverse of the 2x2 diagonal block at dA.
// This is a batch operation where each thread is doing one multiplication.
__global__ void
magmablas_csscal_inverseblock_upper(
    const magmaFloatComplex *dA, int ldda, 
    magmaFloatComplex *dB, int lddb)
{
    int tx  = threadIdx.x;
    
    magmaFloatComplex AKM1K = *dA(0,1);
    magmaFloatComplex AKM1  = MAGMA_C_DIV(*dA(0,0), AKM1K);
    magmaFloatComplex AK    = MAGMA_C_DIV(*dA(1,1), MAGMA_C_CONJ( AKM1K ));
    magmaFloatComplex DENOM = AKM1*AK - MAGMA_C_ONE;

    magmaFloatComplex  BKM1 = MAGMA_C_DIV( *dB(0,tx), AKM1K);
    magmaFloatComplex  BK   = MAGMA_C_DIV( *dB(1,tx), MAGMA_C_CONJ(AKM1K) );

    *dB(0,tx) = MAGMA_C_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(1,tx) = MAGMA_C_DIV( AKM1*BK-BKM1,  DENOM );
}

__global__ void
magmablas_csscal_inverseblock_lower(
    const magmaFloatComplex *dA, int ldda,
    magmaFloatComplex *dB, int lddb)
{
    int tx  = threadIdx.x;

    magmaFloatComplex AKM1K = *dA(1,0);
    magmaFloatComplex AKM1  = MAGMA_C_DIV(*dA(0,0), MAGMA_C_CONJ( AKM1K ) );
    magmaFloatComplex AK    = MAGMA_C_DIV(*dA(1,1), AKM1K );
    magmaFloatComplex DENOM = AKM1*AK - MAGMA_C_ONE;

    magmaFloatComplex  BKM1 = MAGMA_C_DIV( *dB(0,tx),  MAGMA_C_CONJ(AKM1K));
    magmaFloatComplex  BK   = MAGMA_C_DIV( *dB(1,tx), AKM1K );

    *dB(0,tx) = MAGMA_C_DIV(  AK*BKM1-BK ,  DENOM );
    *dB(1,tx) = MAGMA_C_DIV( AKM1*BK-BKM1,  DENOM );
}


/***************************************************************************//**
    Purpose
    -------
    CHETRS solves a system of linear equations dA*dX = dB with a complex
    Hermitian matrix dA using the factorization dA = dU*dD*dU**H or
    dA = dL*dD*dL**H computed by CHETRF_GPU.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t 
            Specifies whether the details of the factorization are stored
            as an upper or lower triangular matrix.
            = MagmaUpper:  Upper triangular, form is A = U*D*U**H;
            = MagmaLower:  Lower triangular, form is A = L*D*L**H.
    
    @param[in]
    n       INTEGER
            The order of the matrix dA.  N >= 0.
    
    @param[in]
    nrhs    INTEGER
            The number of right hand sides, i.e., the number of columns
            of the matrix dB.  NRHS >= 0.
    
    @param[in]
    dA      COMPLEX array, dimension (LDA,N)
            The block diagonal matrix D and the multipliers used to
            obtain the factor U or L as computed by CHETRF_GPU.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,N).
    
    @param[in]
    ipiv    INTEGER array, dimension (N)
            Details of the interchanges and the block structure of D
            as determined by CHETRF.
    
    @param[in,out]
    dB      COMPLEX array, dimension (LDDB, NRHS)
            On entry, the right hand side matrix dB.
            On exit, the solution matrix dX.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,N).
    
    @param[out]
    info    INTEGER
            = 0:  successful exit
            < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_hetrs
*******************************************************************************/
extern "C" magma_int_t
magma_chetrs_gpu(
    magma_uplo_t uplo, magma_int_t n, magma_int_t nrhs,
    magmaFloatComplex *dA, magma_int_t ldda,
    magma_int_t *ipiv,
    magmaFloatComplex *dB, magma_int_t lddb,
    magma_int_t *info, 
    magma_queue_t queue )
{
    /* Constants */
    const magmaFloatComplex c_one     = MAGMA_C_ONE;
    const magmaFloatComplex c_neg_one = MAGMA_C_NEG_ONE;

    /* Local variables */
    int k, kp;
    bool upper = (uplo == MagmaUpper);

    /* Test the input parameters. */
    *info = 0;
    if ( ! upper && uplo != MagmaLower ) {
        *info = -1;
    } else if ( n < 0 ) {
        *info = -2;
    } else if ( nrhs < 0 ) {
        *info = -3;
    } else if ( ldda < max( 1, n ) ) {
        *info = -5;
    } else if ( lddb < max( 1, n ) ) {
        *info = -8;
    }
    
    if ( *info != 0 ) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Quick return if possible */
    if (n == 0 || nrhs == 0) {
        return *info;
    }
    
    if (upper) {
        /* Solve A*X = B, where A = U*D*U'.
           First solve U*D*X = B, overwriting B with X.
           K is the main loop index, decreasing from N to 1 in steps of
           1 or 2, depending on the size of the diagonal blocks.      */
        for(k = n-1; k > -1; k--) {
            if ( ipiv[k] > 0 ) {
                /* 1 x 1 diagonal block.
                   Interchange rows k and ipiv(k). */
                kp = ipiv[k]-1;
                if ( kp != k )
                    magma_cswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);

                /* Multiply by inv(U(K)), where U(K) is the transformation
                   stored in column K of A. */
                magma_cgeru(k, nrhs, c_neg_one, dA(0,k), 1, dB(k,0), lddb, dB(0,0), lddb, queue);

                /* Multiply by the inverse of the diagonal block. */
                magmablas_csscal_inverse<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), dB(k,0), lddb);
            } 
            else {
                /* 2 x 2 diagonal block
                   Interchange rows K-1 and -IPIV(K). */
                kp = -ipiv[k]-1;
                if ( kp != k-1 )
                    magma_cswap(nrhs, dB(k-1,0), lddb, dB(kp,0), lddb, queue);

                /* Multiply by inv(U(K)), where U(K) is the transformation
                   stored in columns K-1 and K of A. */
                magma_cgeru(k-1, nrhs, c_neg_one, dA(0,k  ), 1, dB(k  ,0), lddb, dB, lddb, queue);
                magma_cgeru(k-1, nrhs, c_neg_one, dA(0,k-1), 1, dB(k-1,0), lddb, dB, lddb, queue);

                /* Multiply by the inverse of the diagonal block. */
                magmablas_csscal_inverseblock_upper<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k-1,k-1), ldda, dB(k-1,0), lddb);

                /* reduce k once more for the 2 x 2 block */
                k--;
            }
        }

        /* Next solve U'*X = B, overwriting B with X.
           K is the main loop index, increasing from 1 to N in steps of
           1 or 2, depending on the size of the diagonal blocks.  */
        for(k=0; k<n; k++) {
            if ( ipiv[k] > 0) {
                /* 1 x 1 diagonal block
                   Multiply by inv(U'(K)), where U(K) is the transformation
                   stored in column K of A. */
                if (k > 0) {
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k), 1, c_one, dB(k, 0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv( nrhs, dB(k,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if( kp != k )
                    magma_cswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);
            }
            else {
                /* 2 x 2 diagonal block
                   Multiply by inv(U'(K+1)), where U(K+1) is the transformation
                   stored in columns K and K+1 of A. */
                if (k > 0) {
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k), 1, c_one, dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif

                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k+1,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, k, nrhs, c_neg_one, dB, lddb,
                                dA(0, k+1), 1, c_one, dB(k+1, 0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k+1,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and -IPIV(K). */
                kp = -ipiv[k]-1;
                if ( kp != k )
                    magma_cswap(nrhs, dB(k, 0), lddb, dB(kp, 0), lddb, queue);
            
                /* increase k one more for the 2 x 2 block */
                k++;
            }
        }    
    } else {
        /*  Solve A*X = B, where A = L*D*L'.
            First solve L*D*X = B, overwriting B with X.
    
            K is the main loop index, increasing from 1 to N in steps of
            1 or 2, depending on the size of the diagonal blocks. */
        for(k=0; k<n; k++) {
            if ( ipiv[k] > 0) {
                /* 1 x 1 diagonal block 
                   Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if (0){
                if ( kp != k )
                    magma_cswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);
                
                /* Multiply by inv(L(K)), where L(K) is the transformation
                   stored in column K of A.   */
                if (k < n-1)
                    magma_cgeru(n-k-1, nrhs, c_neg_one, dA(k+1, k), 1, dB(k,0), lddb,
                                dB(k+1, 0), lddb, queue);
                
                /* Multiply by the inverse of the diagonal block. */
                magmablas_csscal_inverse<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), dB(k,0), lddb);
                }     
                else {
                    magma_cgeru_1<<<magma_ceildiv(n-k-1,64), 64, 0, queue->hip_stream()>>>
                        (n, k, kp, nrhs, dA(0,k), dB, lddb);
                    magma_cswap_scal<<<1, nrhs, 0, queue->hip_stream()>>>
                        (k, kp, nrhs, dA(0,k), dB, lddb);
                }
            }
            else {
                /*  2 x 2 diagonal block
                    Interchange rows K+1 and -IPIV(K). */
                kp = -ipiv[k]-1;
                if (0) {
                if ( kp != k+1 )
                    magma_cswap(nrhs, dB(k+1,0), lddb, dB(kp,0), lddb, queue);

                /* Multiply by inv(L(K)), where L(K) is the transformation
                   stored in columns K and K+1 of A. */
                if ( k < n-2 ) {
                    magma_cgeru(n-k-2, nrhs, c_neg_one, dA(k+2,k), 1, dB(k,0), lddb,
                                dB(k+2,0), lddb, queue);
                    magma_cgeru(n-k-2, nrhs, c_neg_one, dA(k+2,k+1), 1, dB(k+1,0), lddb, 
                                dB(k+2,0), lddb, queue);
                }
            
                /* Multiply by the inverse of the diagonal block. */
                magmablas_csscal_inverseblock_lower<<<1, nrhs, 0, queue->hip_stream()>>>
                    (dA(k,k), ldda, dB(k,0), lddb);
                }
                else {
                    magma_cgeru_2<<<magma_ceildiv(n-k-2,64), 64, 0, queue->hip_stream()>>>
                        (n, k, kp, nrhs, dA(0,k), ldda, dB, lddb);
                    magma_cswap_scal_inverseblock_lower<<<1, nrhs, 0, queue->hip_stream()>>>
                        (k, kp, nrhs, dA(k,k), ldda, dB(0, 0), lddb);
                }

                /* increase k one more for the 2 x 2 block */
                k++;
            }
        }
        
        /* Next solve L'*X = B, overwriting B with X.
           K is the main loop index, decreasing from N to 1 in steps of
           1 or 2, depending on the size of the diagonal blocks. */
        for(k = n-1; k > -1; k--) {
            if ( ipiv[k] > 0 ) {
                /* 1 x 1 diagonal block.
                   Multiply by inv(L'(K)), where L(K) is the transformation
                   stored in column K of A. */
                if (1){
                if ( k < n-1 ) {
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k), 1, c_one,
                                dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and IPIV(K). */
                kp = ipiv[k]-1;
                if ( kp!=k )
                    magma_cswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);
                }
                else {
                }
            }
            else {
                /*  2 x 2 diagonal block
                    Multiply by inv(L'(K-1)), where L(K-1) is the transformation
                    stored in columns K-1 and K of A. */
                if ( k < n-1 ) {
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k), 1, c_one,
                                dB(k,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k,0), lddb, queue);
                    #endif

                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k-1,0), lddb, queue);
                    #endif
                    magma_cgemv(MagmaConjTrans, n-k-1, nrhs, c_neg_one,
                                dB(k+1,0), lddb, dA(k+1,k-1), 1, c_one,
                                dB(k-1,0), lddb, queue);
                    #ifdef COMPLEX
                    magmablas_clacgv(nrhs, dB(k-1,0), lddb, queue);
                    #endif
                }

                /* Interchange rows K and -IPIV(K).*/
                kp = -ipiv[k]-1;
                if ( kp != k )
                    magma_cswap(nrhs, dB(k,0), lddb, dB(kp,0), lddb, queue);

                /* reduce k once more for the 2 x 2 block */
                k--;
            }
        }

    }
    return *info;
}
