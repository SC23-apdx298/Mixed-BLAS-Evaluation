#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date February 2023

       @author Azzam Haidar

       @generated from magmablas/zgemv_batched_smallsq.cu, normal z -> s, Wed Feb 22 15:21:25 2023
*/

#include "magma_internal.h"
#include "magma_templates.h"
#include "batched_kernel_param.h"

#define PRECISION_s

#define SLDA(N)              ( (N==15||N==23||N==31)? N : (N+1) )

#ifdef MAGMA_HAVE_CUDA
#define MAX_THREADS          (128)
#else
#define MAX_THREADS          (256)
#endif
#define BATCH_GEMV_NTCOL(N)  (max(1,MAX_THREADS/N))

////////////////////////////////////////////////////////////////////////////////
template<typename T, int N>
__global__ __launch_bounds__(N*BATCH_GEMV_NTCOL(N)) void
sgemvn_batched_smallsq_kernel(
        const T alpha,
        T const * const * dA_array, const T* dA, int ldda, int strideA,
        T const * const * dx_array, const T* dx, int incx, int stridex,
        const T beta,
        T**       dy_array,       T* dy, int incy, int stridey,
        const int batchCount)
{
    extern __shared__ T zdata[];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;

    const int batchid = bx * blockDim.y + ty;
    if(batchid >= batchCount) return;

    const T* A = (dA_array == NULL) ? dA + batchid * strideA : dA_array[batchid];
    const T* x = (dx_array == NULL) ? dx + batchid * stridex : dx_array[batchid];
    T* y = (dy_array == NULL) ? dy + batchid * stridey : dy_array[batchid];

    T rA[N] = {MAGMA_S_ZERO};

    // shared memory
    T* sx = (T*)(zdata);
    sx += ty * N;

    // read x in shmem
    sx[tx] = x[tx * incx];
    __syncthreads();

    T ry = (beta == MAGMA_S_ZERO) ? MAGMA_S_ZERO : beta * y[tx * incy];
    #pragma unroll
    for(int j = 0; j < N; j++) {
        rA[j] = A[j * ldda + tx];
    }

    T rTmp = MAGMA_S_ZERO;
    #pragma unroll
    for(int j = 0; j < N; j++) {
        rTmp += rA[j] * sx[j];
    }

    rTmp *= alpha;
    ry   += rTmp;
    y[tx * incy] = ry;

}

////////////////////////////////////////////////////////////////////////////////
template<typename T, int N>
__global__ __launch_bounds__(N*BATCH_GEMV_NTCOL(N)) void
sgemvc_batched_smallsq_kernel(
        const magma_trans_t transA, const T alpha,
        T const * const * dA_array, const T* dA, int ldda, int strideA,
        T const * const * dx_array, const T* dx, int incx, int stridex,
        const T beta,
        T**      dy_array,       T* dy, int incy, int stridey,
        const int batchCount)
{
    extern __shared__ T zdata[];

    const int tx   = threadIdx.x;
    const int ty   = threadIdx.y;
    const int bx   = blockIdx.x;
    const int nty  = blockDim.y;
    const int slda = SLDA(N);

    const int batchid = bx * blockDim.y + ty;
    if(batchid >= batchCount) return;

    const T* A = (dA_array == NULL) ? dA + batchid * strideA : dA_array[batchid];
    const T* x = (dx_array == NULL) ? dx + batchid * stridex : dx_array[batchid];
    T* y = (dy_array == NULL) ? dy + batchid * stridey : dy_array[batchid];

    T rA[N] = {MAGMA_S_ZERO};

    // shared memory
    T* sA = (T*)(zdata);
    T* sx = sA + nty * slda * N;
    sA += ty * slda * N;
    sx += ty * N;

    T ry = (beta == MAGMA_S_ZERO) ? MAGMA_S_ZERO : beta * y[tx * incy];

    // read x in shmem
    sx[tx] = x[tx * incx];

    #pragma unroll
    for(int j = 0; j < N; j++) {
        rA[j] = A[j * ldda + tx];
    }

    // transpose
    #pragma unroll
    for(int j = 0; j < N; j++) {
        #if defined(PRECISION_z) || defined(PRECISION_c)
        sA[tx * slda + j] = (transA == MagmaConjTrans) ? MAGMA_S_CONJ(rA[j]) : rA[j];
        #else
        sA[tx * slda + j] = rA[j];
        #endif
    }
    __syncthreads();

    #pragma unroll
    for(int j = 0; j < N; j++) {
         rA[j] = sA[j * slda + tx];
    }

    T rTmp = MAGMA_S_ZERO;
    #pragma unroll
    for(int j = 0; j < N; j++) {
        rTmp += rA[j] * sx[j];
    }

    rTmp *= alpha;
    ry   += rTmp;
    y[tx * incy] = ry;
}

////////////////////////////////////////////////////////////////////////////////
template<int N>
static int
sgemv_batched_smallsq_kernel_driver(
    magma_trans_t transA,
    const float alpha,
    float const * const * dA_array, const float* dA, magma_int_t ldda, magma_int_t strideA,
    float const * const * dx_array, const float* dx, magma_int_t incx, magma_int_t stridex,
    const float beta,
    float**      dy_array,       float* dy, magma_int_t incy, magma_int_t stridey,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_device_t device;
    magma_getdevice( &device );
    magma_int_t ntcol  = BATCH_GEMV_NTCOL(N);
    magma_int_t shmem  = N * sizeof(float);
    if( !(transA == MagmaNoTrans) ) {
        const int slda = SLDA(N);
        shmem += slda * N * sizeof(float);
    }
    shmem *= ntcol;

    // get max. dynamic shared memory on the GPU
    int shmem_max, nthreads_max;
    hipDeviceGetAttribute (&nthreads_max, hipDeviceAttributeMaxThreadsPerBlock, device);
    #if CUDA_VERSION >= 9000
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeSharedMemPerBlockOptin, device);
    if (shmem <= shmem_max) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(sgemvn_batched_smallsq_kernel<float), N>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
        hipFuncSetAttribute(reinterpret_cast<const void*>(sgemvc_batched_smallsq_kernel<float), N>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem);
    }
    #else
    hipDeviceGetAttribute (&shmem_max, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    #endif    // CUDA_VERSION >= 9000

    magma_int_t total_threads = N * ntcol;
    if ( total_threads > nthreads_max || shmem > shmem_max ) {
        return -100;
    }

    const int nblocks = magma_ceildiv(batchCount, ntcol);
    dim3 grid(nblocks, 1, 1);
    dim3 threads(N, ntcol, 1);
    if( transA == MagmaNoTrans ) {
        void *kernel_args[] = {
                (void*)&alpha, (void*)&dA_array, (void*)&dA, &ldda, &strideA,
                               (void*)&dx_array, (void*)&dx, &incx, &stridex,
                (void*)&beta,         &dy_array,        &dy, &incy, &stridey, &batchCount};
        hipError_t e = hipLaunchKernel((void*)sgemvn_batched_smallsq_kernel<float, N>, grid, threads, kernel_args, shmem, queue->hip_stream());
        if( e != hipSuccess ) {
            return -100;
        }
    }
    else {
        void *kernel_args[] = {
                &transA,
                (void*)&alpha, (void*)&dA_array, (void*)&dA, &ldda, &strideA,
                               (void*)&dx_array, (void*)&dx, &incx, &stridex,
                (void*)&beta,         &dy_array,        &dy, &incy, &stridey, &batchCount};
        hipError_t e = hipLaunchKernel((void*)sgemvc_batched_smallsq_kernel<float, N>, grid, threads, kernel_args, shmem, queue->hip_stream());
        if( e != hipSuccess ) {
            return -100;
        }
    }

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
static magma_int_t
sgemv_batched_smallsq_core(
    magma_trans_t transA, magma_int_t n,
    const float alpha,
    float const * const * dA_array, const float* dA, magma_int_t ldda, magma_int_t strideA,
    float const * const * dx_array, const float* dx, magma_int_t incx, magma_int_t stridex,
    const float beta,
    float**      dy_array,       float* dy, magma_int_t incy, magma_int_t stridey,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    switch(n){
        case  1: info = sgemv_batched_smallsq_kernel_driver< 1>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  2: info = sgemv_batched_smallsq_kernel_driver< 2>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  3: info = sgemv_batched_smallsq_kernel_driver< 3>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  4: info = sgemv_batched_smallsq_kernel_driver< 4>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  5: info = sgemv_batched_smallsq_kernel_driver< 5>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  6: info = sgemv_batched_smallsq_kernel_driver< 6>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  7: info = sgemv_batched_smallsq_kernel_driver< 7>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  8: info = sgemv_batched_smallsq_kernel_driver< 8>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case  9: info = sgemv_batched_smallsq_kernel_driver< 9>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 10: info = sgemv_batched_smallsq_kernel_driver<10>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 11: info = sgemv_batched_smallsq_kernel_driver<11>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 12: info = sgemv_batched_smallsq_kernel_driver<12>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 13: info = sgemv_batched_smallsq_kernel_driver<13>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 14: info = sgemv_batched_smallsq_kernel_driver<14>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 15: info = sgemv_batched_smallsq_kernel_driver<15>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 16: info = sgemv_batched_smallsq_kernel_driver<16>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 17: info = sgemv_batched_smallsq_kernel_driver<17>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 18: info = sgemv_batched_smallsq_kernel_driver<18>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 19: info = sgemv_batched_smallsq_kernel_driver<19>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 20: info = sgemv_batched_smallsq_kernel_driver<20>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 21: info = sgemv_batched_smallsq_kernel_driver<21>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 22: info = sgemv_batched_smallsq_kernel_driver<22>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 23: info = sgemv_batched_smallsq_kernel_driver<23>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 24: info = sgemv_batched_smallsq_kernel_driver<24>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 25: info = sgemv_batched_smallsq_kernel_driver<25>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 26: info = sgemv_batched_smallsq_kernel_driver<26>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 27: info = sgemv_batched_smallsq_kernel_driver<27>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 28: info = sgemv_batched_smallsq_kernel_driver<28>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 29: info = sgemv_batched_smallsq_kernel_driver<29>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 30: info = sgemv_batched_smallsq_kernel_driver<30>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 31: info = sgemv_batched_smallsq_kernel_driver<31>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        case 32: info = sgemv_batched_smallsq_kernel_driver<32>(transA, alpha, dA_array, dA, ldda, strideA, dx_array, dx, incx, stridex, beta, dy_array, dy, incy, stridey, batchCount, queue); break;
        default:;
    }

    return info;
}

/******************************************************************************/
extern "C" magma_int_t
magmablas_sgemv_batched_smallsq(
    magma_trans_t transA, magma_int_t n,
    const float alpha,
    float const * const * dA_array, magma_int_t ldda,
    float const * const * dx_array, magma_int_t incx,
    const float beta,
    float**      dy_array, magma_int_t incy,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < n )
        info = -5;
    else if ( incx <= 0 )
        info = -7;
    else if ( incy <= 0 )
        info = -10;

    if (info != 0) {
        return info;
    }

    info = sgemv_batched_smallsq_core(
            transA, n,
            alpha, dA_array, NULL, ldda, 0,
                   dx_array, NULL, incx, 0,
            beta,  dy_array, NULL, incy, 0,
            batchCount, queue );

    return info;
}

/******************************************************************************/
extern "C" magma_int_t
magmablas_sgemv_batched_strided_smallsq(
    magma_trans_t transA, magma_int_t n,
    const float alpha,
    const float* dA, magma_int_t ldda, magma_int_t strideA,
    const float* dx, magma_int_t incx, magma_int_t stridex,
    const float beta,
    float* dy, magma_int_t incy, magma_int_t stridey,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < n )
        info = -5;
    else if ( incx <= 0 )
        info = -7;
    else if ( incy <= 0 )
        info = -10;

    if (info != 0) {
        return info;
    }

    info = sgemv_batched_smallsq_core(
            transA, n,
            alpha, NULL, dA, ldda, strideA,
                   NULL, dx, incx, stridex,
            beta,  NULL, dy, incy, stridey,
            batchCount, queue );

    return info;
}
