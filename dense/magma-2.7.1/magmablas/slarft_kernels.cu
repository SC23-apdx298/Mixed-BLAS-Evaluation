#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.7.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date February 2023

       @generated from magmablas/zlarft_kernels.cu, normal z -> s, Wed Feb 22 15:21:16 2023
       @author Azzam Haidar
*/

#include "magma_internal.h"
#include "magma_templates.h"

#define use_gemm_larft

/******************************************************************************/
static  __device__ void
slarft_strmv_sm32x32_device(
    int n, int k, float *tau,
    float *Tin, int ldtin,  float *Tout, int ldtout )
{
    extern __shared__ float shared_data[];

    int tx = threadIdx.x;
    float *sdata = (float*)shared_data;
    float res;

    // this routine apply a sequence of trmv to update k column of the triangular
    // T starting at n-k to n where T is of size n by n and where the first n-k
    // columns of T are supposed updated previously.
    // So the routine load all of T nxn to the shared memory
    // and apply the sequence of trmv.
    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate
    // one element of the column of T then move to the next column

    // read T into shared
    for (int s=0; s < n-k; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }

#if defined(use_gemm_larft)
    for (int s=n-k; s < n; s++)
    {
        if (tx == s)
            sdata[tx + s*n] = tau[s];
        else
            sdata[tx + s*n] = -tau[s] * Tin[tx + s * ldtin];
    }
#else
    for (int s=n-k; s < n; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
#endif

    // perform trmv
    for (int i=n-k; i < n; i++)
    {
        __syncthreads();
        res = MAGMA_S_ZERO;
        if (tx < i)
        {
            for (int j=tx; j < i; j++)
            {
                res += sdata[tx + j * n] * sdata[j+ i * n];
            }
        }
        __syncthreads();
        if (tx < i)
        {
            sdata[tx + i * n] = res;
        }
    }

    __syncthreads();
    // write back the updated block of k column of T
    for (int s=n-k; s < n; s++)
    {
        Tout[tx + s * ldtout] = sdata[tx + s*n];
    }
}


/******************************************************************************/
__global__ void
slarft_strmv_sm32x32_kernel(
    int n, int k, float *tau,
    float *Tin, int ldtin,  float *Tout, int ldtout )
{
    slarft_strmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}


/******************************************************************************/
__global__ void
slarft_strmv_sm32x32_kernel_batched(
    int n, int k,
    float **tau_array, int taui,
    float **Tin_array,  int Tini, int Tinj, int ldtin,
    float **Tout_array, int Touti, int Toutj, int ldtout )
{
    int batchId = blockIdx.z;
    float *tau  = tau_array[batchId]  + taui;
    float *Tin  = Tin_array[batchId]  + Tinj  * ldtin + Tini;
    float *Tout = Tout_array[batchId] + Toutj * ldtout + Touti;
    slarft_strmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}


/******************************************************************************/
extern "C"
void magmablas_slarft_strmv_sm32x32(
    magma_int_t m, magma_int_t n,
    float *tau,
    float *Tin, magma_int_t ldtin,
    float *Tout, magma_int_t ldtout,
    magma_queue_t queue )
{
    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    size_t shmem = sizeof(float)*(m*m);
    slarft_strmv_sm32x32_kernel
        <<< grid, threads, shmem, queue->hip_stream() >>>
        (m, n,  tau, Tin, ldtin, Tout, ldtout);
}


/******************************************************************************/
extern "C"
void magmablas_slarft_strmv_sm32x32_batched(
    magma_int_t m, magma_int_t n,
    float **tau_array, magma_int_t taui,
    float **Tin_array, magma_int_t Tini, magma_int_t Tinj, magma_int_t ldtin,
    float **Tout_array, magma_int_t Touti, magma_int_t Toutj, magma_int_t ldtout,
    magma_int_t batchCount, magma_queue_t queue)
{

    magma_int_t max_batchCount = queue->get_maxBatch();
    dim3 threads(max(m,1), 1, 1);
    size_t shmem = sizeof(float)*(m*m);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid(1, 1, ibatch);

        slarft_strmv_sm32x32_kernel_batched
        <<< grid, threads, shmem, queue->hip_stream() >>>
        (m, n,  tau_array+i, taui,
         Tin_array+i,  Tini,  Tinj,  ldtin,
         Tout_array+i, Touti, Toutj, ldtout);
    }
}


/******************************************************************************/
static __device__ void
slarft_recstrmv_sm32x32_device(
    int m, int n, float *tau,
    float *Trec, int ldtrec, float *Ttri, int ldttri)
{
    extern __shared__ float shared_data[];

    int tx = threadIdx.x;
    float *sdata = (float*)shared_data;
    float res;

    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate
    // one element of the column of T then move to the next column

    // read T into shared
    for (int s=0; s < n; s++)
    {
        sdata[tx + s*n] = Trec[tx + s * ldtrec];
    }
    __syncthreads();

    // perform sequence of n-1 gemv
    for (int i=0; i < n; i++)
    {
        res = MAGMA_S_ZERO;
        for (int j=0; j < i; j++)
        {
            res += sdata[tx + j * n] * Ttri[j+ i * ldttri];
        }
        __syncthreads();   // a enlever
        sdata[tx + i * n] = -tau[i] * (sdata[tx + i * n] + res);
        __syncthreads();
    }

    // write back the updated block of k column of T  multiplying by -tau
    for (int s=0; s < n; s++)
    {
        Trec[tx + s * ldtrec] = sdata[tx + s*n];
    }
}


/******************************************************************************/
__global__ void
slarft_recstrmv_sm32x32_kernel(
    int m, int n, float *tau,
    float *Trec, int ldtrec, float *Ttri, int ldttri)
{
    slarft_recstrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}


/******************************************************************************/
__global__ void
slarft_recstrmv_sm32x32_kernel_batched(
    int m, int n,
    float **tau_array, int taui,
    float **Trec_array, int Treci, int Trecj, int ldtrec,
    float **Ttri_array, int Ttrii, int Ttrij, int ldttri)
{
    int batchId = blockIdx.z;
    float *tau  = tau_array[batchId]  + taui;
    float *Trec = Trec_array[batchId] + Trecj * ldtrec + Treci;
    float *Ttri = Ttri_array[batchId] + Ttrij * ldttri + Ttrii;
    slarft_recstrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}


/******************************************************************************/
extern "C"
void magmablas_slarft_recstrmv_sm32x32(
    magma_int_t m, magma_int_t n,
    float *tau,
    float *Trec, magma_int_t ldtrec,
    float *Ttri, magma_int_t ldttri,
    magma_queue_t queue )
{
    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    size_t shmem = sizeof(float)*(m*n);
    slarft_recstrmv_sm32x32_kernel
        <<< grid, threads, shmem, queue->hip_stream() >>>
        (m, n,  tau, Trec, ldtrec, Ttri, ldttri);
}


/******************************************************************************/
extern "C"
void magmablas_slarft_recstrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n,
    float **tau_array, magma_int_t taui,
    float **Trec_array, magma_int_t Treci, magma_int_t Trecj, magma_int_t ldtrec,
    float **Ttri_array, magma_int_t Ttrii, magma_int_t Ttrij, magma_int_t ldttri,
    magma_int_t batchCount, magma_queue_t queue)
{
    dim3 threads(max(m,1), 1, 1);
    magma_int_t max_batchCount = queue->get_maxBatch();
    size_t shmem = sizeof(float)*(m*n);

    for(magma_int_t i = 0; i < batchCount; i+=max_batchCount) {
        magma_int_t ibatch = min(max_batchCount, batchCount-i);
        dim3 grid(1, 1, ibatch);

        slarft_recstrmv_sm32x32_kernel_batched
        <<< grid, threads, shmem, queue->hip_stream() >>>
        ( m, n,  tau_array+i, taui,
          Trec_array+i, Treci, Trecj, ldtrec,
          Ttri_array+i, Ttrii, Ttrij, ldttri);

    }
}
