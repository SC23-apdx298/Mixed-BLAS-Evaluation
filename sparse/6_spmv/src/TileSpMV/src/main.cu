#include "hip/hip_runtime.h"
#include "biio.h"
#include "power_nvml.hpp"

#include "common.h"
#include "utils.h"
#include "csr2tile.h"
#include "external/CSR5_cuda/anonymouslib_cuda.h"
#include "tilespmv_cpu.h"
#include "tilespmv_cuda.h"

# define INDEX_DATA_TYPE unsigned char


#define DEBUG_FORMATCOST 0


int main(int argc, char ** argv)
{

	if (argc < 2)
    {
        INFO_LOG("Run the code by './test matrix.mtx'.\n");
        return 0;
    }
	
    INFO_LOG("--------------------------------!!!!!!!!------------------------------------\n");

 	struct timeval t1, t2;
	int rowA;
	int colA;
	MAT_PTR_TYPE nnzA;
	int isSymmetricA;
    MAT_VAL_TYPE *csrValA;
    int *csrColIdxA;
    MAT_PTR_TYPE *csrRowPtrA;
    double *csrValA_tmp;
	
    int device_id = atoi(argv[1]);
    // "Usage: ``./spmv -d 0 mtx A.mtx'' for Ax=y on device 0"

    INFO_LOG("device_id = %i\n", device_id);

    char  *filename;
    filename = argv[2];
    INFO_LOG("MAT: -------------- %s --------------\n", filename);

    // load mtx A data to the csr format
    gettimeofday(&t1, NULL);
    read_Dmatrix_32(&rowA, &colA, &nnzA, &csrRowPtrA, &csrColIdxA, &csrValA_tmp, &isSymmetricA, filename);
    gettimeofday(&t2, NULL);
    double time_loadmat  = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    INFO_LOG("  input matrix A: ( %i, %i ) nnz = %i\n  loadfile time    = %4.5f sec\n", rowA, colA, nnzA, time_loadmat/1000.0);

    csrValA = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * nnzA);
    for (size_t i = 0; i < nnzA; i++)
    {
        csrValA[i] = (MAT_VAL_TYPE)csrValA_tmp[i];
    }
    free(csrValA_tmp);

	for (int i = 0; i < nnzA; i++)
	    csrValA[i] = i % 10;

    rowA = (rowA / BLOCK_SIZE) * BLOCK_SIZE;

    // set device
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    INFO_LOG("---------------------------------------------------------------------------------------------\n");
    INFO_LOG("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);



    Tile_matrix *matrixA = (Tile_matrix *)malloc(sizeof (Tile_matrix));

    //format conversion

    Tile_create(matrixA, 
                rowA, colA, nnzA,
                csrRowPtrA,
                csrColIdxA,
                csrValA);

	MAT_VAL_TYPE *x = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * colA);
	for (int i = 0; i < colA; i++)
	{
		x[i] = i % 10;
	}

    // compute reference results on a cpu core

	MAT_VAL_TYPE *y_golden = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
	for (int i = 0; i < rowA; i++)
	{
		MAT_VAL_TYPE sum = 0;
		for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
		{
			sum += csrValA[j] * x[csrColIdxA[j]];
		}
		y_golden[i] = sum;
	}




    //run CPU TileSpMV
    

    struct timeval cpu_tstart,cpu_tend;
    gettimeofday(&cpu_tstart, NULL);

	MAT_VAL_TYPE *y = (MAT_VAL_TYPE *)malloc(sizeof(MAT_VAL_TYPE) * rowA);
    memset(y, 0, sizeof(MAT_VAL_TYPE) * rowA);

    int tilenum = matrixA->tilenum;


    int * ptroffset1 = (int *)malloc(sizeof(int) * tilenum);
    int * ptroffset2 = (int *)malloc(sizeof(int) * tilenum);
    memset(ptroffset1, 0, sizeof(int) * tilenum);
    memset(ptroffset2, 0, sizeof(int) * tilenum);

    int rowblkblock = 0;

    unsigned int * blkcoostylerowidx ;
    int * blkcoostylerowidx_colstart   ;
    int * blkcoostylerowidx_colstop ;
    int *multicoo_ptr = (int *)malloc((rowA + 1) * sizeof(int));

    int *multicoo_colidx ;
    MAT_VAL_TYPE *multicoo_val ;

    tilespmv_cpu(matrixA,
                ptroffset1,
                ptroffset2,
                &rowblkblock,
                &blkcoostylerowidx,
                &blkcoostylerowidx_colstart,
                &blkcoostylerowidx_colstop,
                rowA, colA, nnzA,
                csrRowPtrA,
                csrColIdxA,
                csrValA,
                x,
                y,
                y_golden
            );


  MAT_VAL_TYPE alpha = 1.0;
  memset(y, 0, sizeof(MAT_VAL_TYPE) * rowA);


//run GPU TilespMV

    call_tilespmv_cuda( filename,
                        matrixA,
                        ptroffset1,
                        ptroffset2,
                        rowblkblock,
                        blkcoostylerowidx,
                        blkcoostylerowidx_colstart,
                        blkcoostylerowidx_colstop,
                        rowA, colA, nnzA,
                        csrRowPtrA,
                        csrColIdxA,
                        csrValA,
                        alpha,
                        x,
                        y,
                        y_golden);



    //check results

    int error_count_cuda = 0;
    for (int i = 0; i < rowA; i++)
        if (abs(y_golden[i] - y[i]) > 0.01 * abs(y[i]))
        {
            error_count_cuda++;
            // cout<<"y_golden = "<<y_golden[i]<<" , "<<"y = "<<y[i]<<endl;
        }

    if (error_count_cuda == 0)
        INFO_LOG("Check... PASS!\n");
    else
        INFO_LOG("Check... NO PASS! error_count_cuda = %d\n", error_count_cuda);

    free(matrixA);
    free(csrValA);
    free(csrColIdxA);
    free(csrRowPtrA);

}
